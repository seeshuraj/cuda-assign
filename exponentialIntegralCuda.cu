#include "hip/hip_runtime.h"
#include "exponentialIntegralCuda.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

#define CHECK(call) { \
    const hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(1); \
    } \
}

// Device constants for maximum float/double values and NaN representations
__constant__ float float_max = 3.402823466e+38f;  // FLT_MAX
__constant__ float float_nan = 0x7fc00000;        // Quiet NaN bit pattern
__constant__ double double_max = 1.7976931348623157e+308;  // DBL_MAX
__constant__ double double_nan = 0x7ff8000000000000;       // Quiet NaN bit pattern

__device__ float exponentialIntegralFloatDevice(int n, float x, int maxIterations) {
    const float eulerConstant = 0.5772156649015329f;
    const float epsilon = 1.E-30f;
    const float bigfloat = float_max;
    const int nm1 = n - 1;
    float ans = 0.0f;

    if (n < 0 || x < 0.0f || (x == 0.0f && (n == 0 || n == 1))) {
        return float_nan;
    }
    
    if (n == 0) {
        ans = expf(-x)/x;
    } else if (x > 1.0f) {
        float b = x + n;
        float c = bigfloat;
        float d = 1.0f/b;
        float h = d;
        
        for (int i = 1; i <= maxIterations; i++) {
            const float a = -i*(nm1 + i);
            b += 2.0f;
            d = 1.0f/(a*d + b);
            c = b + a/c;
            const float del = c*d;
            h *= del;
            if (fabsf(del - 1.0f) <= epsilon) {
                return h*expf(-x);
            }
        }
        return h*expf(-x);
    } else {
        ans = (nm1 != 0) ? 1.0f/nm1 : -logf(x)-eulerConstant;
        float fact = 1.0f;
        
        for (int i = 1; i <= maxIterations; i++) {
            fact *= -x/i;
            float del;
            
            if (i != nm1) {
                del = -fact/(i-nm1);
            } else {
                float psi = -eulerConstant;
                for (int ii = 1; ii <= nm1; ii++) {
                    psi += 1.0f/ii;
                }
                del = fact*(-logf(x) + psi);
            }
            
            ans += del;
            if (fabsf(del) < fabsf(ans)*epsilon) return ans;
        }
    }
    return ans;
}

__device__ double exponentialIntegralDoubleDevice(int n, double x, int maxIterations) {
    const double eulerConstant = 0.5772156649015329;
    const double epsilon = 1.E-30;
    const double bigDouble = double_max;
    const int nm1 = n - 1;
    double ans = 0.0;

    if (n < 0 || x < 0.0 || (x == 0.0 && (n == 0 || n == 1))) {
        return double_nan;
    }
    
    if (n == 0) {
        ans = exp(-x)/x;
    } else if (x > 1.0) {
        double b = x + n;
        double c = bigDouble;
        double d = 1.0/b;
        double h = d;
        
        for (int i = 1; i <= maxIterations; i++) {
            const double a = -i*(nm1 + i);
            b += 2.0;
            d = 1.0/(a*d + b);
            c = b + a/c;
            const double del = c*d;
            h *= del;
            if (fabs(del - 1.0) <= epsilon) {
                return h*exp(-x);
            }
        }
        return h*exp(-x);
    } else {
        ans = (nm1 != 0) ? 1.0/nm1 : -log(x)-eulerConstant;
        double fact = 1.0;
        
        for (int i = 1; i <= maxIterations; i++) {
            fact *= -x/i;
            double del;
            
            if (i != nm1) {
                del = -fact/(i-nm1);
            } else {
                double psi = -eulerConstant;
                for (int ii = 1; ii <= nm1; ii++) {
                    psi += 1.0/ii;
                }
                del = fact*(-log(x) + psi);
            }
            
            ans += del;
            if (fabs(del) < fabs(ans)*epsilon) return ans;
        }
    }
    return ans;
}

__global__ void exponentialIntegralFloatKernel(float* results, int max_n, int max_samples, 
                                             double a, double b, int maxIterations) {
    const int order = blockIdx.y*blockDim.y + threadIdx.y + 1;
    const int sampleIdx = blockIdx.x*blockDim.x + threadIdx.x;

    if (order <= max_n && sampleIdx < max_samples) {
        const double x_val = a + (sampleIdx + 1)*((b - a)/max_samples);
        const float x = static_cast<float>(x_val);
        results[(order-1)*max_samples + sampleIdx] = exponentialIntegralFloatDevice(order, x, maxIterations);
    }
}

__global__ void exponentialIntegralDoubleKernel(double* results, int max_n, int max_samples, 
                                              double a, double b, int maxIterations) {
    const int order = blockIdx.y*blockDim.y + threadIdx.y + 1;
    const int sampleIdx = blockIdx.x*blockDim.x + threadIdx.x;

    if (order <= max_n && sampleIdx < max_samples) {
        const double x = a + (sampleIdx + 1)*((b - a)/max_samples);
        results[(order-1)*max_samples + sampleIdx] = exponentialIntegralDoubleDevice(order, x, maxIterations);
    }
}

void runGPU(std::vector<std::vector<float>>& resultsFloatGpu,
            std::vector<std::vector<double>>& resultsDoubleGpu,
            unsigned int n, unsigned int numberOfSamples,
            double a, double b, int maxIterations,
            double& timeTotalGpu,
            double& timeFloatKernel,
            double& timeDoubleKernel,
            bool verbose) {
    float* d_results_float = nullptr;
    double* d_results_double = nullptr;
    float* h_results_float = new float[n * numberOfSamples];
    double* h_results_double = new double[n * numberOfSamples];

    // Warm-up GPU to avoid initialization timing artifacts
    hipFree(0);

    CHECK(hipMalloc((void**)&d_results_float, n * numberOfSamples * sizeof(float)));
    CHECK(hipMalloc((void**)&d_results_double, n * numberOfSamples * sizeof(double)));

    // Configure kernel launch parameters
    dim3 block(16, 16);
    dim3 grid((numberOfSamples + block.x - 1)/block.x, 
              (n + block.y - 1)/block.y);

    if (verbose) {
        std::cout << "GPU Kernel Configuration:" << std::endl;
        std::cout << "  Grid size: (" << grid.x << ", " << grid.y << ")" << std::endl;
        std::cout << "  Block size: (" << block.x << ", " << block.y << ")" << std::endl;
        std::cout << "  Total threads: " << grid.x*grid.y*block.x*block.y << std::endl;
    }

    hipEvent_t start, stop, startFloat, stopFloat, startDouble, stopDouble;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));
    CHECK(hipEventCreate(&startFloat));
    CHECK(hipEventCreate(&stopFloat));
    CHECK(hipEventCreate(&startDouble));
    CHECK(hipEventCreate(&stopDouble));

    CHECK(hipEventRecord(start));

    // Launch float precision kernel
    CHECK(hipEventRecord(startFloat));
    exponentialIntegralFloatKernel<<<grid, block>>>(d_results_float, n, numberOfSamples, a, b, maxIterations);
    CHECK(hipGetLastError());
    CHECK(hipEventRecord(stopFloat));
    
    // Launch double precision kernel
    CHECK(hipEventRecord(startDouble));
    exponentialIntegralDoubleKernel<<<grid, block>>>(d_results_double, n, numberOfSamples, a, b, maxIterations);
    CHECK(hipGetLastError());
    CHECK(hipEventRecord(stopDouble));

    // Copy results back to host
    CHECK(hipMemcpy(h_results_float, d_results_float, n * numberOfSamples * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(h_results_double, d_results_double, n * numberOfSamples * sizeof(double), hipMemcpyDeviceToHost));
    
    CHECK(hipEventRecord(stop));
    CHECK(hipEventSynchronize(stop));

    // Get timing results
    float tempTime;
    CHECK(hipEventElapsedTime(&tempTime, startFloat, stopFloat));
    timeFloatKernel = tempTime/1000.0;
    
    CHECK(hipEventElapsedTime(&tempTime, startDouble, stopDouble));
    timeDoubleKernel = tempTime/1000.0;
    
    CHECK(hipEventElapsedTime(&tempTime, start, stop));
    timeTotalGpu = tempTime/1000.0;

    // Copy results to 2D vectors
    for (unsigned int i = 0; i < n; i++) {
        for (unsigned int j = 0; j < numberOfSamples; j++) {
            resultsFloatGpu[i][j] = h_results_float[i * numberOfSamples + j];
            resultsDoubleGpu[i][j] = h_results_double[i * numberOfSamples + j];
        }
    }

    // Cleanup
    delete[] h_results_float;
    delete[] h_results_double;
    CHECK(hipFree(d_results_float));
    CHECK(hipFree(d_results_double));
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));
    CHECK(hipEventDestroy(startFloat));
    CHECK(hipEventDestroy(stopFloat));
    CHECK(hipEventDestroy(startDouble));
    CHECK(hipEventDestroy(stopDouble));
}
